#include "hip/hip_runtime.h"
/* Suma de matrices por hilos */


#include <ctime>
#include <iostream>
#include <time.h>
#include <math.h>
#define N 90000000

using namespace std;

void generateRandom(int *h_a);
void parallelAddition();
void serialAddition();
// en el host "h_"
// en device "d_"
int *h_a, *h_b, *h_c, *serialC;
int *d_a, *d_b, *d_c;
int size = N * sizeof(int);
double serialTimer;
float parallelTimer;
// definición del Kernel.
__global__ void vectorAdd(int *d_a, int *d_b, int *d_c){
    // múltiples bloques e hilos se deben de generar.
    // se necesita un índice para mapear...
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    d_c[index] = d_a[index] + d_b[index];
}

int main(int argc, char const *argv[])
{
    h_a = (int * ) malloc(size);
    h_b = (int * ) malloc(size);
    h_c = (int * ) malloc(size);
    serialC = (int * ) malloc(size);
    
    generateRandom(h_a);
    generateRandom(h_b);

    parallelAddition();
    serialAddition();
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    cout << "Speedup: " << (serialTimer / parallelTimer) << endl;
}

void generateRandom(int *h_a){
    srand(time(NULL));
    for (int i = 0; i < N; i++){
        h_a[i] = rand() % 101;
    }
}

void parallelAddition(){
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    // transferir datos de host a device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threads = 512;
    int blocks  = ceil(N/threads);

    // eventos para tomar tiempo
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // llamada a kernel, número de bloques y de kernel
    vectorAdd<<<blocks, threads>>>(d_a, d_b, d_c);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    hipEventElapsedTime(&parallelTimer, start, stop);

    cout << "ellapser parallel timer: " << parallelTimer << "ms" << endl;

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
}

void serialAddition(){
    clock_t start = clock();

    for(int i = 0; i<N; ++i){
        h_c[i] = h_a[i] + h_b[i];
    }
    clock_t end = clock();
    serialTimer = double(end-start) / double(CLOCKS_PER_SEC);
    cout << "Elapsed Serial Time: " << serialTimer << endl;
}